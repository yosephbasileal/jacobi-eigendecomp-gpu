
/* jacobi.c: Cyclic Jacobi method for finding eigenvalues and eigenvectrors
 * 
 * Author: Basileal Imana
 * Date: 06/10/16
 */

// Libriaries

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <getopt.h>
#include <stdbool.h>
//#include <cublas_v2.h>
//#include <cuda_runtime.h>

/** Cuda handle error, if err is not success print error and line in code
*
* @param status CUDA Error types
*/
#define HANDLE_ERROR(status) \
{ \
   if (status != hipSuccess) \
   { \
      fprintf(stderr, "%s failed  at line %d \nError message: %s \n", \
         __FILE__, __LINE__ ,hipGetErrorString(status)); \
      exit(EXIT_FAILURE); \
   } \
}

bool debug = false; // -d command line option for verbose output



// Prints a matrix to stdout
void print(float* A, int size) {
   for(int i = 0; i < size; i++) {
      for(int j = 0; j < size; j++) {
         printf("%.4f  ",A [i*size +j]);
      }
      printf("\n");
   }
}

// Prints a non-square matrix to stdout
void print2(float* A, int row, int col) {
   for(int i = 0; i < row; i++) {
      for(int j = 0; j < col; j++) {
         printf("%.4f  ", A[i*col+j]);
      }
      printf("\n");
   }
}

// Copies matrix elements 'from' to 'to'
void copy(float* from, float* to, int size) {
   for(int i = 0; i < size; i++) {
      for(int j = 0; j < size; j++) {
         to[i*size+j] = from[i*size+j];
      }
   }
}

// Makes matrix A an identity matrix
void eye(float* A, int size) {
   for(int i = 0; i < size; i++) {
      for(int j = i; j < size; j++) {
         if(i == j) {
            A[i*size+j] = 1.0;
         } else {
            A[i*size+j] = 0.0;
            A[j*size+i] = 0.0;
         }
      }
   }
}

// Make all non diagonal elements 0
void remove_nondiag(float* A, int size) {
   for(int i = 0; i < size; i++) {
      for(int j = 0; j < size; j++) {
         if(i != j) {
            A[i*size+j] = 0.0;
         }
      }
   }
}

// Checks if matrix is symmetric
bool is_symmetric(float* A, int size) {
   for(int i = 0; i < size - 1; i++) {
      for(int j = i + 1; j < size; j++) {
         if(A[i*size+j] != A[j*size+i]) {
            return false;
         }
      }
   }
   return true;
}

// Calculates the square root of sum of squares of
// all off diagonal elements of A
float off(float* A, int size) {
   float sum = 0;
   for(int i = 0; i < size - 1; i++) {
      for(int j = i + 1; j < size; j++) {
         // multiply by 2 to account for other half of matrix
         sum += 2 * A[i*size+j] * A[i*size+j];
      }
   }

   return sqrt(sum);
}


// Initalizes arrays for chess tournament ordering
void chess_initialize(int* order1, int* order2, int size) {
   int curr = -1;
	for(int i = 0; i < size; i++) {
		order1[i] = ++curr;
      order2[i] = ++curr;
   }
}

// Do one permutation of chess tournament ordering
void chess_permute(int* order1, int* order2, int size) {
   // save the first element of array 2
   int temp = order2[0];
   // shift everthing in array 2 to the left
   for(int i = 0; i <= size - 2; i++) {
      order2[i] = order2[i+1];
   }
   // put last element of array 1 as last element array 2
   order2[size-1] = order1[size-1];
   // shift everything but the first two of array 1 to the right
   for(int i = size - 1; i >= 2; i--) {
      order1[i] = order1[i-1];
   }
   // put first element of array 2 as second element of array 1
   order1[1] = temp;
}

// Swaps two numbers
__host__ __device__ void swap(int* num1, int* num2) {
   int temp = *num1;
   *num1 = *num2;
   *num2 = temp;
}

// Given pivot(i,j), constructs a submatrix of rows affected J'*A
__device__ void create_sub_row(float* A, int size, int i, int j, float* A_sub) {
   for(int k = 0; k < size; k++) {
      A_sub[0 * size + k] = A[i * size + k];
      A_sub[1 * size + k] = A[j * size + k];
   }
}

// Given pivot(i,j), constructs a submatrix of row affected by A*J
__device__ void create_sub_col(float* A, int size, int i, int j, float* A_sub) {
   for(int k = 0; k < size; k++) {
      A_sub[k * 2 + 0] = A[k * size + i];
      A_sub[k * 2 + 1] = A[k * size + j];
   }
}

// Updates the original matrix's rows with changes made to submatrix
__device__ void update_sub_row(float* A, int size, int i, int j, float* A_sub) {
   for(int k = 0; k < size; k++) {
      A[i * size + k] = A_sub[0 * size + k];
      A[j * size + k] = A_sub[1 * size + k];
   }
}

// Updates the original matrix's cols with changes made to submatrix
__device__ void update_sub_col(float* A, int size, int i, int j, float* A_sub) {
   for(int k = 0; k < size; k++) {
      A[k * size + i] = A_sub[k * 2 + 0];
      A[k * size + j] = A_sub[k * 2 + 1];
   }
}

// Cacluates values of c and s for a given pivot of rotation (i,j)
void jacobi_cs(float* A, int size, int i, int j, float* c, float* s) {
   // calculate T
   float T = (A[j*size+j] - A[i*size+i]) / (2 * A[i*size+j]);

   // equation: t^2 + 2Tt - 1 = 0
   // chose the root that is smaller in absolute value
   float t;
   if(T >= 0) {
      t = -T + sqrt(1.0 + T*T);
   } else {
      t = -T - sqrt(1.0 + T*T);
   }

   // calculate c and s
   *c = 1.0 / (sqrt(1.0 + t*t));
   *s = *c * t;
}


// Multiplies A(mxk) matrix by B(kxn) matrix
__device__ void mul_mat(int m,int n,int k, float* a,float* b, float* c)
{
    int i,j,h;
    for(i = 0; i < m; i++)
    {
        for(j = 0; j < n; j++)
        {
             c[i * n + j] = 0;
             for(h = 0; h< k; h++)
               c[i * n + j] += + a[i * k + h] * b[h * n + j];
        }
    }
}


__global__ void jacobi_kernel1(float* D_d, float* E_d, int size, int* arr1, int* arr2, float* cc, float* ss) {

	// get i,j pair
	int i = arr1[threadIdx.x];
	int j = arr2[threadIdx.x];

	// make sure i < j
	if(i > j) {
		swap(&i, &j);
	}

	// get precaculated values of c and s for current values of i and j
	float c = cc[threadIdx.x];
	float s = ss[threadIdx.x];

	// setup rotation matrix
	float R_t[] = {c, -s, s, c};

	// Submatrices (2xn or nx2 size) for storing intermediate results
   float* D_sub = (float *) malloc(sizeof(float) * 2 * size);
   float* E_sub = (float *) malloc(sizeof(float) * 2 * size);
   float* X_sub = (float *) malloc(sizeof(float) * 2 * size);

	// get submatrix of rows of D that will be affected by R' * D
   create_sub_row(D_d, size, i, j, D_sub);

   // sgemm calculates C = alpha*A*B + beta*C
   //float alp = 1.0;
   //float bet = 0.0;
	//const float* alpha = &alp;
	//const float* beta = &bet;

	// Create a handle for CUBLAS
	//cublasHandle_t handle;
	//cublasCreate(&handle);

   // calculate X_sub = R' * D_sub
   //cublasSgemm(handle,CUBLAS_OP_T, CUBLAS_OP_N, \
               2, size, 2, alpha, R, 2, D_sub, size, beta, X_sub, size); 
	mul_mat(2,size,2,R_t,D_sub,X_sub);

	// Destroy CUBLAS handle
	//cublasDestroy(handle);

   // update D
   update_sub_row(D_d,size,i,j,X_sub);

	// free memory
	free(D_sub);
	free(E_sub);
	free(X_sub);
}


__global__ void jacobi_kernel2(float* D_d, float* E_d, int size, int* arr1, int* arr2, float* cc, float* ss) {
	
	// get i,j pair
	int i = arr1[threadIdx.x];
	int j = arr2[threadIdx.x];

	// make sure i < j
	if(i > j) {
		swap(&i,&j);
	}

	// get precaculated values of c and s for current values of i and j
	float c = cc[threadIdx.x];
   float s = ss[threadIdx.x];

   // setup rotation matrix
   float R[] = {c, s, -s, c};

   // Submatrices (2xn or nx2 size) for storing intermediate results
   float* D_sub = (float *) malloc(sizeof(float) * 2 * size);
   float* E_sub = (float *) malloc(sizeof(float) * 2 * size);
   float* X_sub = (float *) malloc(sizeof(float) * 2 * size);

	// get submatrix of cols of D that will be affected by D * R
   create_sub_col(D_d,size,i,j,D_sub);

	// sgemm calculate C = alpha*A*B + beta*C
	//float alp = 1.0;
	//float bet = 0.0;
   //const float* alpha = &alp;
   //const float* beta = &bet;


   // create a handle for CUBLAS
	//cublasHandle_t handle;
	//cublasCreate(&handle);

	// calculate X_sub = D_sub * R
   //cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, \
               size, 2, 2, alpha, D_sub, size, R, 2, beta, X_sub, size);
	 mul_mat(size,2,2,D_sub,R,X_sub);

   // update D
   update_sub_col(D_d,size,i,j,X_sub);

   // get submatrix of cols of E that iwll be affected by E * R
   create_sub_col(E_d,size,i,j,E_sub);

   // calculate X_sub = E_sub * R
   //cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, \
                    size, 2, 2, alpha, E_sub, size, R, 2, beta, X_sub, size);

	mul_mat(size,2,2,E_sub,R,X_sub);

	//cublasDestroy(handle);

   // update E
   update_sub_col(E_d,size,i,j,X_sub);

	// free memory
	free(D_sub);
	free(E_sub);
	free(X_sub);
}

// Jacobi method
void jacobi(float* A, float* D, float* E, int size, float epsilon) {
   // initialize D and E
   copy(A, D, size);
   eye(E, size);

   // device memory pointers
   float *D_d, *E_d;

	// chess tournament ordering
	int *arr1, *arr2;
	
	// store c and s values for each permutation
	float *cc, *ss;

	hipError_t cudaStatus;

	// allocate unified memory
	hipMallocManaged(&arr1, sizeof(int) * size/2);
	hipMallocManaged(&arr2, sizeof(int) * size/2);
	hipMallocManaged(&cc, sizeof(float) * size/2);
	hipMallocManaged(&ss, sizeof(float) * size/2);	
   hipMallocManaged(&D_d, sizeof(float) * size*size);

	// allocate device memory
   hipMalloc((void **) &E_d, sizeof(float) * size*size);

	// copy matrices to device
	copy(D,D_d,size);
   hipMemcpy(E_d, E, sizeof(float) * size*size, hipMemcpyHostToDevice);
	
	// do sweeps
	while(off(D_d,size) > epsilon) {
		// initialize ordering of i,j pairs
		chess_initialize(arr1, arr2, size/2);
		
		for(int h = 0; h < size-1; h++) {

			// precalcuate values of c and s for current permuationt so
			// that both kernels use the same rotation matrix
			for(int k = 0; k < size/2; k++) {
				int i = arr1[k];
				int j = arr2[k];
				if(i > j) swap(&i,&j);
				jacobi_cs(D_d, size, i, j,&cc[k],&ss[k]);
			}

			// launch kernel 1
			jacobi_kernel1<<<1,size/2>>>(D_d, E_d, size, arr1, arr2, cc,ss);
		
			// synchronize
			cudaStatus = hipDeviceSynchronize();	
			HANDLE_ERROR(cudaStatus);

			// launch kernel 2
			jacobi_kernel2<<<1,size/2>>>(D_d, E_d, size, arr1, arr2, cc, ss);
			
			// synchronize
         cudaStatus = hipDeviceSynchronize();
         HANDLE_ERROR(cudaStatus);

			// do next permutation of i, j pairs
			chess_permute(arr1, arr2, size/2);
		}

		if(debug) {
			printf("One sweep done. New matrix D: \n");
			print(D_d, size);
			printf("\n");
		}
   }
	
	// copy to host
	copy(D_d,D,size);
	hipMemcpy(E, E_d, sizeof(float) * size*size, hipMemcpyDeviceToHost);

	// free memory
	hipFree(arr1);
	hipFree(arr2);
	hipFree(D_d);
	hipFree(E_d);
}

// Main
int main(int argc, char** argv) {

   // process command line arguments
   int r;
   while ((r = getopt(argc, argv, "d")) != -1) {
      switch(r)
      {
         case 'd':
            debug = true;
            break;
         default:
            exit(1);
      }
   }

   // read matrix size from stdin
   int size;
   scanf("%d",&size);

   // initialize array
   float* A = (float*) malloc(sizeof(float) * size * size);
   float* D = (float*) malloc(sizeof(float) * size * size);
   float* E = (float*) malloc(sizeof(float) * size * size);

   // read matrix from stdin
   for(int i = 0; i < size; i++) {
      for(int j = 0; j < size; j++) {
         scanf("%f", &A[i * size + j]);
      }
   }

   // make sure matrix is symmetric
   if(!is_symmetric(A, size)) {
      printf("Error: Given matrix not symmetric!\n");
      return 0;
   }
 
   if(debug) {
      printf("Input matrix A: \n");
      print(A, size);
      printf("\n");
   }

   // desired accuracy
   float epsilon = 0.01;

   // call facobi method
   jacobi(A, D, E, size, epsilon);
   remove_nondiag(D, size);

   // output results
   printf("\n");
   printf("______Results______\n");
   printf("Eigenvalues on the diagonal:\n");
   print(D, size);
   printf("\n");
   printf("Corresponding eigenvectors:\n");
   print(E, size);
   printf("\n");

	// clean up
	free(A);
	free(D);
	free(E);

   return 0;
}

