#include "hip/hip_runtime.h"

/* Cyclic Jacobi method for finding eigenvalues and eigenvectrors
 * of real symmetric matrices
 *
 * Author: Basileal Imana
 * Date: 06/10/16
 */

// Libriaries
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <getopt.h>
#include <stdbool.h>
#include <time.h>
#include "utils.h"

/** Cuda handle error, if err is not success print error and line in code
*
* @param status CUDA Error types
*/
#define HANDLE_ERROR(status) \
{ \
   if (status != hipSuccess) \
   { \
      fprintf(stderr, "%s failed  at line %d \nError message: %s \n", \
         __FILE__, __LINE__ ,hipGetErrorString(status)); \
      exit(EXIT_FAILURE); \
   } \
}

bool debug = false; // -d command line option for verbose output
bool output = false; // -p command line option for ouputting results

double epsilon = 0.01; // -e command line option for desired accuracy
int num_sweeps = 6; // -s command line option for number of sweeps

// Initalizes arrays for chess tournament ordering
void chess_initialize(int* order1, int* order2, int size) {
   int curr = -1;
   for(int i = 0; i < size; i++) {
     order1[i] = ++curr;
     order2[i] = ++curr;
   }
}

// Do one permutation of chess tournament ordering
void chess_permute(int* order1, int* order2, int size) {
   // save the first element of array 2
   int temp = order2[0];
   // shift everthing in array 2 to the left
   for(int i = 0; i <= size - 2; i++) {
      order2[i] = order2[i+1];
   }
   // put last element of array 1 as last element array 2
   order2[size-1] = order1[size-1];
   // shift everything but the first two of array 1 to the right
   for(int i = size - 1; i >= 2; i--) {
      order1[i] = order1[i-1];
   }
   // put first element of array 2 as second element of array 1
   order1[1] = temp;
}

// Cacluates values of c and s for a given pivot of rotation (i,j)
void jacobi_cs(double* A, int size, int i, int j, double* c, double* s) {
   // calculate T
   double T = (A[j*size+j] - A[i*size+i]) / (2 * A[i*size+j]);

   // equation: t^2 + 2Tt - 1 = 0
   // chose the root that is smaller in absolute value
   double t;
   if(T >= 0) {
      t = -T + sqrt(1.0 + T*T);
   } else {
      t = -T - sqrt(1.0 + T*T);
   }

   // calculate c and s
   *c = 1.0 / (sqrt(1.0 + t*t));
   *s = *c * t;
}

__global__ void jacobi_kernel1(double* D, double* X, int size, int* arr1, int* arr2, double* cc, double* ss) {

   int tid = threadIdx.x;

   // get i,j pair, all threads in block operate on row i and row j
   int i = arr1[blockIdx.x];
   int j = arr2[blockIdx.x];

   // make sure i < j
   if(i > j) {
      int temp = i;
      i = j;
      j = temp;
   }

   // get precaculated values of c and s for current values of i and j
   double c = cc[blockIdx.x];
   double s = ss[blockIdx.x];

   // setup rotation matrix
   double R_T[] = {c, -s, s, c};

   // get row i and row j elements for current thread
   double row_i = D[i*size+tid];
   double row_j = D[j*size+tid];

   // calculate X = R' * D, X is column major array
   X[tid*size+i] = R_T[0] * row_i + R_T[1] * row_j;
   X[tid*size+j] = R_T[2] * row_i + R_T[3] * row_j;
}


__global__ void jacobi_kernel2(double* D, double* E, double* X, int size, int* arr1, int* arr2, double* cc, double* ss) {

   int tid = threadIdx.x;

   // get i,j pair, all threads in block operate on col i and col j
   int i = arr1[blockIdx.x];
   int j = arr2[blockIdx.x];

   // make sure i < j
   if(i > j) {
      int temp = i;
      i = j;
      j = temp;
   }

   // get precaculated values of c and s for current values of i and j
   double c = cc[blockIdx.x];
   double s = ss[blockIdx.x];

   // setup rotation matrix
   double R[] = {c, s, -s, c};

   // get col i and col j elements of X for current thread
   double x_col_i = X[i*size+tid];
   double x_col_j = X[j*size+tid];

   // calculate D = X * R, X is column major array
   D[i*size+tid] = x_col_i * R[0] + x_col_j * R[2];
   D[j*size+tid] = x_col_i * R[1] + x_col_j * R[3];

   // get col i and col j elements of E for current thread
   double e_col_i = E[i*size+tid];
   double e_col_j = E[j*size+tid];

   // caclulate E = E * R, E is column major array
   E[i*size+tid] = e_col_i * R[0] + e_col_j * R[2];
   E[j*size+tid] = e_col_i * R[1] + e_col_j * R[3];
}

// Jacobi method
void jacobi(double* A, double* D, double* E, int size, double epsilon, int num_sweeps) {
   printf("Initializing jacobi matrices...\n");

   // initialize D and E
   copy(A, D, size);
   eye(E, size);

   // device memory pointers for matrices
   double *D_d, *E_d, *X_d; //E and X are column major arrays

   // chess tournament ordering arr1 stores i, arr2 stroes j
   int *arr1, *arr2;

   // store c and s values for corresponding (i,j) pair
   double *cc, *ss;

   hipError_t cudaStatus;

   // allocate unified memory
   hipMallocManaged(&arr1, sizeof(int) * size/2);
   hipMallocManaged(&arr2, sizeof(int) * size/2);
   hipMallocManaged(&cc, sizeof(double) * size/2);
   hipMallocManaged(&ss, sizeof(double) * size/2);
   hipMallocManaged(&D_d, sizeof(double) * size*size);

   // allocate device memory
   hipMalloc((void **) &E_d, sizeof(double) * size*size);
   hipMalloc((void **) &X_d, sizeof(double) * size*size);

   // copy matrices to device
   copy(D,D_d,size);
   hipMemcpy(E_d, E, sizeof(double) * size*size, hipMemcpyHostToDevice);

   int sweep_count = 0;
   double offA;

   // do sweeps
   while((offA = off(D_d,size)) > epsilon && (sweep_count < num_sweeps)) {

      sweep_count++;
      printf("Doing sweep #%d  off(D) = %.8lf \n", sweep_count, offA);

      // initialize ordering of i,j pairs
      chess_initialize(arr1, arr2, size/2);

      for(int h = 0; h < size-1; h++) {

         // precalcuate values of c and s for current permuationt so
         // that both kernels use the same rotation matrix
         for(int k = 0; k < size/2; k++) {
            int i = arr1[k];
            int j = arr2[k];
            if(i > j) {
               int temp = i;
               i = j;
               j = temp;
                                }
            jacobi_cs(D_d, size, i, j,&cc[k],&ss[k]);
         }

         // launch kernel 1
         jacobi_kernel1<<<size/2,size>>>(D_d, X_d, size, arr1, arr2, cc,ss);

         // synchronize
         cudaStatus = hipDeviceSynchronize();
         HANDLE_ERROR(cudaStatus);

         // launch kernel 2
         jacobi_kernel2<<<size/2,size>>>(D_d, E_d, X_d, size, arr1, arr2, cc, ss);

         // synchronize
         cudaStatus = hipDeviceSynchronize();
         HANDLE_ERROR(cudaStatus);

         // do next permutation of i, j pairs
         chess_permute(arr1, arr2, size/2);
      }

      if(debug) {
         printf("One sweep done. New matrix D: \n");
         print(D_d, size);
         printf("\n");
      }
   }

   // copy to host
   copy(D_d,D,size);
   hipMemcpy(E, E_d, sizeof(double) * size*size, hipMemcpyDeviceToHost);

   // free memory
   hipFree(arr1);
   hipFree(arr2);
   hipFree(cc);
   hipFree(ss);
   hipFree(D_d);
   hipFree(E_d);
   hipFree(X_d);
}

// Main
int main(int argc, char** argv) {

   // process command line arguments
   int r;
   while ((r = getopt(argc, argv, "dps:e:")) != -1) {
      switch(r)
      {
         case 'd':
            debug = true;
            break;
         case 'p':
            output = true;
            break;
         case 's':
            num_sweeps = atoi(optarg);
            break;
         case 'e':
            epsilon = atof(optarg);
            break;
         default:
            exit(1);
      }
   }

   printf("Reading matrix from file...\n");
   // read matrix size from stdin
   int size;
   scanf("%d",&size);

   // initialize arrays
   double* A = (double*) malloc(sizeof(double) * size * size);
   double* D = (double*) malloc(sizeof(double) * size * size);
   double* E = (double*) malloc(sizeof(double) * size * size);

   // array to store eigenvalues
   double* ei = (double *) malloc(sizeof(double) * size);

   // read matrix from stdin
   for(int i = 0; i < size; i++) {
      for(int j = 0; j < size; j++) {
         scanf("%lf", &A[i * size + j]);
      }
   }

   // make sure matrix is symmetric
   if(!is_symmetric(A, size)) {
      printf("Warning: Given matrix not symmetric!\n");
      //return 0;
   }

   if(debug) {
      printf("Input matrix A: \n");
      print(A, size);
      printf("\n");
   }

   clock_t begin, end;
   double time_spent;

   begin = clock();

   // call facobi method
   jacobi(A, D, E, size, epsilon, num_sweeps);

   end = clock();
   time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

   printf("Post-processing...\n");
   remove_nondiag(D, size);
   get_diagonals(ei, D, size);
   qsort(ei, size, sizeof(double), compare);

   // output results
   if(output) {
      printf("\n");
      printf("Sorted Eigenvalues:\n");
      for(int i = 0; i < size; i++) {
         printf("%.15lf\n",ei[i]);
      }
      printf("\n");
      //printf("Eigenvectors:\n");
      //print_cm(E, size);
      //printf("\n");
   }

   printf("Execution time of Jacobi: %lf\n", time_spent);

   // clean up
   free(A);
   free(D);
   free(E);

   return 0;
}

